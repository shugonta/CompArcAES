#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

__constant__ int rkey[44];
__constant__ unsigned char SboxCUDA[256] = {
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};

__device__ int mul3CUDA(unsigned char dt) {
  int x;
  x = dt << 1;
  if (x & 0x100)
    x = (x ^ 0x1b) & 0xff;
  x ^= dt;

  return (x);
}

__device__ int mul2CUDA(unsigned char dt) {
  int x;
  x = dt << 1;
  if (x & 0x100)
    x = (x ^ 0x1b) & 0xff;

  return (x);
}

__device__ void CipherCUDA(int *pt, unsigned char *ct, int *rkey) {
  int rnd, threadId = ((blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  int *state = pt;
  unsigned char cb[NBb2];
  int *cw = (int *) cb;

  cw[0] = state[0] ^ rkey[0];
  cw[1] = state[1] ^ rkey[1];
  cw[2] = state[2] ^ rkey[2];
  cw[3] = state[3] ^ rkey[3];
//round 1
  cw[4] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
           SboxCUDA[((unsigned char *) cw)[10]] ^
           SboxCUDA[((unsigned char *) cw)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            SboxCUDA[((unsigned char *) cw)[15]] ^
            SboxCUDA[((unsigned char *) cw)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            SboxCUDA[((unsigned char *) cw)[0]] ^
            SboxCUDA[((unsigned char *) cw)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            SboxCUDA[((unsigned char *) cw)[5]] ^
            SboxCUDA[((unsigned char *) cw)[10]]) << 24)
          ^ rkey[4];

  cw[5] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
           SboxCUDA[((unsigned char *) cw)[14]] ^
           SboxCUDA[((unsigned char *) cw)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            SboxCUDA[((unsigned char *) cw)[3]] ^
            SboxCUDA[((unsigned char *) cw)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            SboxCUDA[((unsigned char *) cw)[4]] ^
            SboxCUDA[((unsigned char *) cw)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            SboxCUDA[((unsigned char *) cw)[9]] ^
            SboxCUDA[((unsigned char *) cw)[14]]) << 24)
          ^ rkey[5];

  cw[6] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
           SboxCUDA[((unsigned char *) cw)[2]] ^
           SboxCUDA[((unsigned char *) cw)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            SboxCUDA[((unsigned char *) cw)[7]] ^
            SboxCUDA[((unsigned char *) cw)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            SboxCUDA[((unsigned char *) cw)[8]] ^
            SboxCUDA[((unsigned char *) cw)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            SboxCUDA[((unsigned char *) cw)[13]] ^
            SboxCUDA[((unsigned char *) cw)[2]]) << 24)
          ^ rkey[6];

  cw[7] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
           SboxCUDA[((unsigned char *) cw)[6]] ^
           SboxCUDA[((unsigned char *) cw)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            SboxCUDA[((unsigned char *) cw)[11]] ^
            SboxCUDA[((unsigned char *) cw)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            SboxCUDA[((unsigned char *) cw)[12]] ^
            SboxCUDA[((unsigned char *) cw)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            SboxCUDA[((unsigned char *) cw)[1]] ^
            SboxCUDA[((unsigned char *) cw)[6]]) << 24)
          ^ rkey[7];
//round 2
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
           SboxCUDA[((unsigned char *) cw)[26]] ^
           SboxCUDA[((unsigned char *) cw)[31]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            SboxCUDA[((unsigned char *) cw)[31]] ^
            SboxCUDA[((unsigned char *) cw)[16]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            SboxCUDA[((unsigned char *) cw)[16]] ^
            SboxCUDA[((unsigned char *) cw)[21]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
            SboxCUDA[((unsigned char *) cw)[21]] ^
            SboxCUDA[((unsigned char *) cw)[26]]) << 24)
          ^ rkey[8];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
           SboxCUDA[((unsigned char *) cw)[30]] ^
           SboxCUDA[((unsigned char *) cw)[19]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            SboxCUDA[((unsigned char *) cw)[19]] ^
            SboxCUDA[((unsigned char *) cw)[20]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            SboxCUDA[((unsigned char *) cw)[20]] ^
            SboxCUDA[((unsigned char *) cw)[25]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
            SboxCUDA[((unsigned char *) cw)[25]] ^
            SboxCUDA[((unsigned char *) cw)[30]]) << 24)
          ^ rkey[9];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
           SboxCUDA[((unsigned char *) cw)[18]] ^
           SboxCUDA[((unsigned char *) cw)[23]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            SboxCUDA[((unsigned char *) cw)[23]] ^
            SboxCUDA[((unsigned char *) cw)[24]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            SboxCUDA[((unsigned char *) cw)[24]] ^
            SboxCUDA[((unsigned char *) cw)[29]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
            SboxCUDA[((unsigned char *) cw)[29]] ^
            SboxCUDA[((unsigned char *) cw)[18]]) << 24)
          ^ rkey[10];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
           SboxCUDA[((unsigned char *) cw)[22]] ^
           SboxCUDA[((unsigned char *) cw)[27]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            SboxCUDA[((unsigned char *) cw)[27]] ^
            SboxCUDA[((unsigned char *) cw)[28]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            SboxCUDA[((unsigned char *) cw)[28]] ^
            SboxCUDA[((unsigned char *) cw)[17]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
            SboxCUDA[((unsigned char *) cw)[17]] ^
            SboxCUDA[((unsigned char *) cw)[22]]) << 24)
          ^ rkey[11];

  //round 3
  cw[4] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
           SboxCUDA[((unsigned char *) cw)[10]] ^
           SboxCUDA[((unsigned char *) cw)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            SboxCUDA[((unsigned char *) cw)[15]] ^
            SboxCUDA[((unsigned char *) cw)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            SboxCUDA[((unsigned char *) cw)[0]] ^
            SboxCUDA[((unsigned char *) cw)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            SboxCUDA[((unsigned char *) cw)[5]] ^
            SboxCUDA[((unsigned char *) cw)[10]]) << 24)
          ^ rkey[12];

  cw[5] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
           SboxCUDA[((unsigned char *) cw)[14]] ^
           SboxCUDA[((unsigned char *) cw)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            SboxCUDA[((unsigned char *) cw)[3]] ^
            SboxCUDA[((unsigned char *) cw)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            SboxCUDA[((unsigned char *) cw)[4]] ^
            SboxCUDA[((unsigned char *) cw)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            SboxCUDA[((unsigned char *) cw)[9]] ^
            SboxCUDA[((unsigned char *) cw)[14]]) << 24)
          ^ rkey[13];

  cw[6] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
           SboxCUDA[((unsigned char *) cw)[2]] ^
           SboxCUDA[((unsigned char *) cw)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            SboxCUDA[((unsigned char *) cw)[7]] ^
            SboxCUDA[((unsigned char *) cw)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            SboxCUDA[((unsigned char *) cw)[8]] ^
            SboxCUDA[((unsigned char *) cw)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            SboxCUDA[((unsigned char *) cw)[13]] ^
            SboxCUDA[((unsigned char *) cw)[2]]) << 24)
          ^ rkey[14];

  cw[7] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
           SboxCUDA[((unsigned char *) cw)[6]] ^
           SboxCUDA[((unsigned char *) cw)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            SboxCUDA[((unsigned char *) cw)[11]] ^
            SboxCUDA[((unsigned char *) cw)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            SboxCUDA[((unsigned char *) cw)[12]] ^
            SboxCUDA[((unsigned char *) cw)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            SboxCUDA[((unsigned char *) cw)[1]] ^
            SboxCUDA[((unsigned char *) cw)[6]]) << 24)
          ^ rkey[15];
//round 4
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
           SboxCUDA[((unsigned char *) cw)[26]] ^
           SboxCUDA[((unsigned char *) cw)[31]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            SboxCUDA[((unsigned char *) cw)[31]] ^
            SboxCUDA[((unsigned char *) cw)[16]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            SboxCUDA[((unsigned char *) cw)[16]] ^
            SboxCUDA[((unsigned char *) cw)[21]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
            SboxCUDA[((unsigned char *) cw)[21]] ^
            SboxCUDA[((unsigned char *) cw)[26]]) << 24)
          ^ rkey[16];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
           SboxCUDA[((unsigned char *) cw)[30]] ^
           SboxCUDA[((unsigned char *) cw)[19]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            SboxCUDA[((unsigned char *) cw)[19]] ^
            SboxCUDA[((unsigned char *) cw)[20]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            SboxCUDA[((unsigned char *) cw)[20]] ^
            SboxCUDA[((unsigned char *) cw)[25]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
            SboxCUDA[((unsigned char *) cw)[25]] ^
            SboxCUDA[((unsigned char *) cw)[30]]) << 24)
          ^ rkey[17];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
           SboxCUDA[((unsigned char *) cw)[18]] ^
           SboxCUDA[((unsigned char *) cw)[23]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            SboxCUDA[((unsigned char *) cw)[23]] ^
            SboxCUDA[((unsigned char *) cw)[24]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            SboxCUDA[((unsigned char *) cw)[24]] ^
            SboxCUDA[((unsigned char *) cw)[29]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
            SboxCUDA[((unsigned char *) cw)[29]] ^
            SboxCUDA[((unsigned char *) cw)[18]]) << 24)
          ^ rkey[18];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
           SboxCUDA[((unsigned char *) cw)[22]] ^
           SboxCUDA[((unsigned char *) cw)[27]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            SboxCUDA[((unsigned char *) cw)[27]] ^
            SboxCUDA[((unsigned char *) cw)[28]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            SboxCUDA[((unsigned char *) cw)[28]] ^
            SboxCUDA[((unsigned char *) cw)[17]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
            SboxCUDA[((unsigned char *) cw)[17]] ^
            SboxCUDA[((unsigned char *) cw)[22]]) << 24)
          ^ rkey[19];

  //round 5
  cw[4] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
           SboxCUDA[((unsigned char *) cw)[10]] ^
           SboxCUDA[((unsigned char *) cw)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            SboxCUDA[((unsigned char *) cw)[15]] ^
            SboxCUDA[((unsigned char *) cw)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            SboxCUDA[((unsigned char *) cw)[0]] ^
            SboxCUDA[((unsigned char *) cw)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            SboxCUDA[((unsigned char *) cw)[5]] ^
            SboxCUDA[((unsigned char *) cw)[10]]) << 24)
          ^ rkey[20];

  cw[5] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
           SboxCUDA[((unsigned char *) cw)[14]] ^
           SboxCUDA[((unsigned char *) cw)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            SboxCUDA[((unsigned char *) cw)[3]] ^
            SboxCUDA[((unsigned char *) cw)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            SboxCUDA[((unsigned char *) cw)[4]] ^
            SboxCUDA[((unsigned char *) cw)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            SboxCUDA[((unsigned char *) cw)[9]] ^
            SboxCUDA[((unsigned char *) cw)[14]]) << 24)
          ^ rkey[21];

  cw[6] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
           SboxCUDA[((unsigned char *) cw)[2]] ^
           SboxCUDA[((unsigned char *) cw)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            SboxCUDA[((unsigned char *) cw)[7]] ^
            SboxCUDA[((unsigned char *) cw)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            SboxCUDA[((unsigned char *) cw)[8]] ^
            SboxCUDA[((unsigned char *) cw)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            SboxCUDA[((unsigned char *) cw)[13]] ^
            SboxCUDA[((unsigned char *) cw)[2]]) << 24)
          ^ rkey[22];

  cw[7] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
           SboxCUDA[((unsigned char *) cw)[6]] ^
           SboxCUDA[((unsigned char *) cw)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            SboxCUDA[((unsigned char *) cw)[11]] ^
            SboxCUDA[((unsigned char *) cw)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            SboxCUDA[((unsigned char *) cw)[12]] ^
            SboxCUDA[((unsigned char *) cw)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            SboxCUDA[((unsigned char *) cw)[1]] ^
            SboxCUDA[((unsigned char *) cw)[6]]) << 24)
          ^ rkey[23];
//round 6
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
           SboxCUDA[((unsigned char *) cw)[26]] ^
           SboxCUDA[((unsigned char *) cw)[31]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            SboxCUDA[((unsigned char *) cw)[31]] ^
            SboxCUDA[((unsigned char *) cw)[16]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            SboxCUDA[((unsigned char *) cw)[16]] ^
            SboxCUDA[((unsigned char *) cw)[21]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
            SboxCUDA[((unsigned char *) cw)[21]] ^
            SboxCUDA[((unsigned char *) cw)[26]]) << 24)
          ^ rkey[24];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
           SboxCUDA[((unsigned char *) cw)[30]] ^
           SboxCUDA[((unsigned char *) cw)[19]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            SboxCUDA[((unsigned char *) cw)[19]] ^
            SboxCUDA[((unsigned char *) cw)[20]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            SboxCUDA[((unsigned char *) cw)[20]] ^
            SboxCUDA[((unsigned char *) cw)[25]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
            SboxCUDA[((unsigned char *) cw)[25]] ^
            SboxCUDA[((unsigned char *) cw)[30]]) << 24)
          ^ rkey[25];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
           SboxCUDA[((unsigned char *) cw)[18]] ^
           SboxCUDA[((unsigned char *) cw)[23]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            SboxCUDA[((unsigned char *) cw)[23]] ^
            SboxCUDA[((unsigned char *) cw)[24]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            SboxCUDA[((unsigned char *) cw)[24]] ^
            SboxCUDA[((unsigned char *) cw)[29]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
            SboxCUDA[((unsigned char *) cw)[29]] ^
            SboxCUDA[((unsigned char *) cw)[18]]) << 24)
          ^ rkey[26];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
           SboxCUDA[((unsigned char *) cw)[22]] ^
           SboxCUDA[((unsigned char *) cw)[27]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            SboxCUDA[((unsigned char *) cw)[27]] ^
            SboxCUDA[((unsigned char *) cw)[28]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            SboxCUDA[((unsigned char *) cw)[28]] ^
            SboxCUDA[((unsigned char *) cw)[17]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
            SboxCUDA[((unsigned char *) cw)[17]] ^
            SboxCUDA[((unsigned char *) cw)[22]]) << 24)
          ^ rkey[27];

  //round 7
  cw[4] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
           SboxCUDA[((unsigned char *) cw)[10]] ^
           SboxCUDA[((unsigned char *) cw)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            SboxCUDA[((unsigned char *) cw)[15]] ^
            SboxCUDA[((unsigned char *) cw)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            SboxCUDA[((unsigned char *) cw)[0]] ^
            SboxCUDA[((unsigned char *) cw)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            SboxCUDA[((unsigned char *) cw)[5]] ^
            SboxCUDA[((unsigned char *) cw)[10]]) << 24)
          ^ rkey[28];

  cw[5] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
           SboxCUDA[((unsigned char *) cw)[14]] ^
           SboxCUDA[((unsigned char *) cw)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            SboxCUDA[((unsigned char *) cw)[3]] ^
            SboxCUDA[((unsigned char *) cw)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            SboxCUDA[((unsigned char *) cw)[4]] ^
            SboxCUDA[((unsigned char *) cw)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            SboxCUDA[((unsigned char *) cw)[9]] ^
            SboxCUDA[((unsigned char *) cw)[14]]) << 24)
          ^ rkey[29];

  cw[6] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
           SboxCUDA[((unsigned char *) cw)[2]] ^
           SboxCUDA[((unsigned char *) cw)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            SboxCUDA[((unsigned char *) cw)[7]] ^
            SboxCUDA[((unsigned char *) cw)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            SboxCUDA[((unsigned char *) cw)[8]] ^
            SboxCUDA[((unsigned char *) cw)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            SboxCUDA[((unsigned char *) cw)[13]] ^
            SboxCUDA[((unsigned char *) cw)[2]]) << 24)
          ^ rkey[30];

  cw[7] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
           SboxCUDA[((unsigned char *) cw)[6]] ^
           SboxCUDA[((unsigned char *) cw)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            SboxCUDA[((unsigned char *) cw)[11]] ^
            SboxCUDA[((unsigned char *) cw)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            SboxCUDA[((unsigned char *) cw)[12]] ^
            SboxCUDA[((unsigned char *) cw)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            SboxCUDA[((unsigned char *) cw)[1]] ^
            SboxCUDA[((unsigned char *) cw)[6]]) << 24)
          ^ rkey[31];
//round 8
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
           SboxCUDA[((unsigned char *) cw)[26]] ^
           SboxCUDA[((unsigned char *) cw)[31]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[21]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            SboxCUDA[((unsigned char *) cw)[31]] ^
            SboxCUDA[((unsigned char *) cw)[16]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[26]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            SboxCUDA[((unsigned char *) cw)[16]] ^
            SboxCUDA[((unsigned char *) cw)[21]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[31]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[16]]) ^
            SboxCUDA[((unsigned char *) cw)[21]] ^
            SboxCUDA[((unsigned char *) cw)[26]]) << 24)
          ^ rkey[32];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
           SboxCUDA[((unsigned char *) cw)[30]] ^
           SboxCUDA[((unsigned char *) cw)[19]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[25]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            SboxCUDA[((unsigned char *) cw)[19]] ^
            SboxCUDA[((unsigned char *) cw)[20]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[30]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            SboxCUDA[((unsigned char *) cw)[20]] ^
            SboxCUDA[((unsigned char *) cw)[25]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[19]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[20]]) ^
            SboxCUDA[((unsigned char *) cw)[25]] ^
            SboxCUDA[((unsigned char *) cw)[30]]) << 24)
          ^ rkey[33];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
           SboxCUDA[((unsigned char *) cw)[18]] ^
           SboxCUDA[((unsigned char *) cw)[23]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[29]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            SboxCUDA[((unsigned char *) cw)[23]] ^
            SboxCUDA[((unsigned char *) cw)[24]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[18]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            SboxCUDA[((unsigned char *) cw)[24]] ^
            SboxCUDA[((unsigned char *) cw)[29]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[23]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[24]]) ^
            SboxCUDA[((unsigned char *) cw)[29]] ^
            SboxCUDA[((unsigned char *) cw)[18]]) << 24)
          ^ rkey[34];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
           SboxCUDA[((unsigned char *) cw)[22]] ^
           SboxCUDA[((unsigned char *) cw)[27]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[17]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            SboxCUDA[((unsigned char *) cw)[27]] ^
            SboxCUDA[((unsigned char *) cw)[28]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[22]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            SboxCUDA[((unsigned char *) cw)[28]] ^
            SboxCUDA[((unsigned char *) cw)[17]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[27]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[28]]) ^
            SboxCUDA[((unsigned char *) cw)[17]] ^
            SboxCUDA[((unsigned char *) cw)[22]]) << 24)
          ^ rkey[35];

  //round 9
  cw[4] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
           SboxCUDA[((unsigned char *) cw)[10]] ^
           SboxCUDA[((unsigned char *) cw)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            SboxCUDA[((unsigned char *) cw)[15]] ^
            SboxCUDA[((unsigned char *) cw)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            SboxCUDA[((unsigned char *) cw)[0]] ^
            SboxCUDA[((unsigned char *) cw)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            SboxCUDA[((unsigned char *) cw)[5]] ^
            SboxCUDA[((unsigned char *) cw)[10]]) << 24)
          ^ rkey[36];

  cw[5] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
           SboxCUDA[((unsigned char *) cw)[14]] ^
           SboxCUDA[((unsigned char *) cw)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            SboxCUDA[((unsigned char *) cw)[3]] ^
            SboxCUDA[((unsigned char *) cw)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            SboxCUDA[((unsigned char *) cw)[4]] ^
            SboxCUDA[((unsigned char *) cw)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            SboxCUDA[((unsigned char *) cw)[9]] ^
            SboxCUDA[((unsigned char *) cw)[14]]) << 24)
          ^ rkey[37];

  cw[6] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
           SboxCUDA[((unsigned char *) cw)[2]] ^
           SboxCUDA[((unsigned char *) cw)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            SboxCUDA[((unsigned char *) cw)[7]] ^
            SboxCUDA[((unsigned char *) cw)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            SboxCUDA[((unsigned char *) cw)[8]] ^
            SboxCUDA[((unsigned char *) cw)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            SboxCUDA[((unsigned char *) cw)[13]] ^
            SboxCUDA[((unsigned char *) cw)[2]]) << 24)
          ^ rkey[38];

  cw[7] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
           SboxCUDA[((unsigned char *) cw)[6]] ^
           SboxCUDA[((unsigned char *) cw)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            SboxCUDA[((unsigned char *) cw)[11]] ^
            SboxCUDA[((unsigned char *) cw)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            SboxCUDA[((unsigned char *) cw)[12]] ^
            SboxCUDA[((unsigned char *) cw)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            SboxCUDA[((unsigned char *) cw)[1]] ^
            SboxCUDA[((unsigned char *) cw)[6]]) << 24)
          ^ rkey[39];
  
 /* for (rnd = 12; rnd < NR4; rnd += 4) {
    cw[index2w] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 0]]) ^
                   mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 5]]) ^
                   SboxCUDA[((unsigned char *) cw)[index | 10]] ^
                   SboxCUDA[((unsigned char *) cw)[index | 15]]
                   |
                   (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 5]]) ^
                    mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 10]]) ^
                    SboxCUDA[((unsigned char *) cw)[index | 15]] ^
                    SboxCUDA[((unsigned char *) cw)[index | 0]]) << 8
                   |
                   (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 10]]) ^
                    mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 15]]) ^
                    SboxCUDA[((unsigned char *) cw)[index | 0]] ^
                    SboxCUDA[((unsigned char *) cw)[index | 5]]) << 16
                   |
                   (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 15]]) ^
                    mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 0]]) ^
                    SboxCUDA[((unsigned char *) cw)[index | 5]] ^
                    SboxCUDA[((unsigned char *) cw)[index | 10]]) << 24)
                  ^ rkey[rnd];

    cw[index2w | 1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 4]]) ^
                       mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 9]]) ^
                       SboxCUDA[((unsigned char *) cw)[index | 14]] ^
                       SboxCUDA[((unsigned char *) cw)[index | 3]]
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 9]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 14]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 3]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 4]]) << 8
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 14]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 3]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 4]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 9]]) << 16
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 3]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 4]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 9]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 14]]) << 24)
                      ^ rkey[rnd | 1];

    cw[index2w | 2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 8]]) ^
                       mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 13]]) ^
                       SboxCUDA[((unsigned char *) cw)[index | 2]] ^
                       SboxCUDA[((unsigned char *) cw)[index | 7]]
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 13]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 2]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 7]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 8]]) << 8
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 2]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 7]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 8]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 13]]) << 16
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 7]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 8]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 13]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 2]]) << 24)
                      ^ rkey[rnd | 2];

    cw[index2w | 3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 12]]) ^
                       mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 1]]) ^
                       SboxCUDA[((unsigned char *) cw)[index | 6]] ^
                       SboxCUDA[((unsigned char *) cw)[index | 11]]
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 1]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 6]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 11]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 12]]) << 8
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 6]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 11]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 12]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 1]]) << 16
                       |
                       (mul2CUDA(SboxCUDA[((unsigned char *) cw)[index | 11]]) ^
                        mul3CUDA(SboxCUDA[((unsigned char *) cw)[index | 12]]) ^
                        SboxCUDA[((unsigned char *) cw)[index | 1]] ^
                        SboxCUDA[((unsigned char *) cw)[index | 6]]) << 24)
                      ^ rkey[rnd | 3];
    unsigned char swap = index;
    index = index2;
    index2 = swap;
    swap = indexw;
    indexw = index2w;
    index2w = swap;
  }*/
  if (threadId == 0) {
    printf("cw0: 0x%x\n", cw[4]);
    printf("cw1: 0x%x\n", cw[5]);
    printf("cw2: 0x%x\n", cw[6]);
    printf("cw3: 0x%x\n", cw[7]);
  }
  cb[0] = SboxCUDA[cb[16]];
  cb[1] = SboxCUDA[cb[21]];
  cb[2] = SboxCUDA[cb[26]];
  cb[3] = SboxCUDA[cb[31]];
  ((int*)ct)[threadId] = cw[0] ^ rkey[40];
  cb[4] = SboxCUDA[cb[20]];
  cb[5] = SboxCUDA[cb[25]];
  cb[6] = SboxCUDA[cb[30]];
  cb[7] = SboxCUDA[cb[19]];
  ((int *) ct)[threadId | 1] = cw[1] ^ rkey[41];
  cb[8] = SboxCUDA[cb[24]];
  cb[9] = SboxCUDA[cb[29]];
  cb[10] = SboxCUDA[cb[18]];
  cb[11] = SboxCUDA[cb[23]];
  ((int *) ct)[threadId | 2] = cw[2] ^ rkey[42];
  cb[12] = SboxCUDA[cb[28]];
  cb[13] = SboxCUDA[cb[17]];
  cb[14] = SboxCUDA[cb[22]];
  cb[15] = SboxCUDA[cb[27]];
  ((int *) ct)[threadId | 3] = cw[3] ^ rkey[43];
  if (threadId == 0) {
    printf("cw0: 0x%x\n", ((int *) ct)[threadId]);
    printf("cw1: 0x%x\n", ((int *) ct)[threadId | 1]);
    printf("cw2: 0x%x\n", ((int *) ct)[threadId | 2]);
    printf("cw3: 0x%x\n", ((int *) ct)[threadId | 3]);
  }
  return;
}

__global__ void device_aes_encrypt(unsigned char *pt, unsigned char *ct, long int size) {

  //This kernel executes AES encryption on a GPU.
  //Please modify this kernel!!
  int thread_id = ((blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

  /* if (thread_id == 0)
     printf("size = %ld\n", size);
 //  printf("You can use printf function to eliminate bugs in your kernel.\n");
 */
//  __shared__ int state[BLOCKSIZE][NB];
//  memcpy(&(state[threadIdx.x][0]), &(pt[thread_id << 4]), sizeof(unsigned char) * NBb);
  CipherCUDA((int *)(&pt[thread_id << 4]), ct, rkey);
//  memcpy(&ct[thread_id << 4], &state[threadIdx.x], sizeof(unsigned char) * NBb);
}

void launch_aes_kernel(unsigned char *pt, int *rk, unsigned char *ct, long int size) {
  //This function launches the AES kernel.
  //Please modify this function for AES kernel.
  //In this function, you need to allocate the device memory and so on.
  unsigned char *d_pt, *d_ct;

  dim3 dim_grid(GRIDSIZE_X, GRIDSIZE_Y, GRIDSIZE_Z), dim_block(BLOCKSIZE, 1, 1);

  hipMalloc((void **) &d_pt, sizeof(unsigned char) * size);
//  hipMalloc((void **) &d_rkey, sizeof(int) * 44);
  hipMalloc((void **) &d_ct, sizeof(unsigned char) * size);

//  hipMemset(d_pt, 0, sizeof(unsigned char) * size);
  hipMemcpy(d_pt, pt, sizeof(unsigned char) * size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(rkey), rk, sizeof(int) * 44);
//  hipMemcpyToSymbol(HIP_SYMBOL(state_org), pt, sizeof(unsigned char) * size);

  device_aes_encrypt <<< dim_grid, dim_block >>> (d_pt, d_ct, size);

  hipMemcpy(ct, d_ct, sizeof(unsigned char) * size, hipMemcpyDeviceToHost);

  hipFree(d_pt);
  hipFree(d_ct);
}












