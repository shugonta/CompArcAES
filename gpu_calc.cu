#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "calculation.h"

__constant__ int rkey[44];
__constant__ unsigned char SboxCUDA[256] = {
        0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
        0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
        0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
        0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
        0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
        0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
        0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
        0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
        0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
        0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
        0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
        0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
        0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
        0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
        0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
        0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};

__device__ void SubShift(int *state){
  unsigned char cb[NBb];
  cb[0] = SboxCUDA[((unsigned char *) state)[0]];
  cb[1] = SboxCUDA[((unsigned char *) state)[5]];
  cb[2] = SboxCUDA[((unsigned char *) state)[10]];
  cb[3] = SboxCUDA[((unsigned char *) state)[15]];
  cb[4] = SboxCUDA[((unsigned char *) state)[4]];
  cb[5] = SboxCUDA[((unsigned char *) state)[9]];
  cb[6] = SboxCUDA[((unsigned char *) state)[14]];
  cb[7] = SboxCUDA[((unsigned char *) state)[3]];
  cb[8] = SboxCUDA[((unsigned char *) state)[8]];
  cb[9] = SboxCUDA[((unsigned char *) state)[13]];
  cb[10] = SboxCUDA[((unsigned char *) state)[2]];
  cb[11] = SboxCUDA[((unsigned char *) state)[7]];
  cb[12] = SboxCUDA[((unsigned char *) state)[12]];
  cb[13] = SboxCUDA[((unsigned char *) state)[1]];
  cb[14] = SboxCUDA[((unsigned char *) state)[6]];
  cb[15] = SboxCUDA[((unsigned char *) state)[11]];
  memcpy(state, cb, sizeof(unsigned char) * NBb);
}
/*
__device__ void SubBytesCUDA(int *state) {
  unsigned char cb[NBb];
  cb[0] = SboxCUDA[((unsigned char *) state)[0]];
  cb[1] = SboxCUDA[((unsigned char *) state)[1]];
  cb[2] = SboxCUDA[((unsigned char *) state)[2]];
  cb[3] = SboxCUDA[((unsigned char *) state)[3]];
  cb[4] = SboxCUDA[((unsigned char *) state)[4]];
  cb[5] = SboxCUDA[((unsigned char *) state)[5]];
  cb[6] = SboxCUDA[((unsigned char *) state)[6]];
  cb[7] = SboxCUDA[((unsigned char *) state)[7]];
  cb[8] = SboxCUDA[((unsigned char *) state)[8]];
  cb[9] = SboxCUDA[((unsigned char *) state)[9]];
  cb[10] = SboxCUDA[((unsigned char *) state)[10]];
  cb[11] = SboxCUDA[((unsigned char *) state)[11]];
  cb[12] = SboxCUDA[((unsigned char *) state)[12]];
  cb[13] = SboxCUDA[((unsigned char *) state)[13]];
  cb[14] = SboxCUDA[((unsigned char *) state)[14]];
  cb[15] = SboxCUDA[((unsigned char *) state)[15]];
  memcpy(state, cb, sizeof(unsigned char) * NBb);
}

__device__ void ShiftRowsCUDA(int *state) {
  unsigned char cb[NBb];

  cb[0] = ((unsigned char *) state)[0];
  cb[1] = ((unsigned char *) state)[5];
  cb[2] = ((unsigned char *) state)[10];
  cb[3] = ((unsigned char *) state)[15];
  cb[4] = ((unsigned char *) state)[4];
  cb[5] = ((unsigned char *) state)[9];
  cb[6] = ((unsigned char *) state)[14];
  cb[7] = ((unsigned char *) state)[3];
  cb[8] = ((unsigned char *) state)[8];
  cb[9] = ((unsigned char *) state)[13];
  cb[10] = ((unsigned char *) state)[2];
  cb[11] = ((unsigned char *) state)[7];
  cb[12] = ((unsigned char *) state)[12];
  cb[13] = ((unsigned char *) state)[1];
  cb[14] = ((unsigned char *) state)[6];
  cb[15] = ((unsigned char *) state)[11];



*//*  cw[0] =
          ((unsigned char *) state)[0] |
          ((unsigned char *) state)[5] << 8 |
          ((unsigned char *) state)[10] << 16 |
          ((unsigned char *) state)[15] << 24;
  cw[1] =
          ((unsigned char *) state)[4] |
          ((unsigned char *) state)[9] << 8 |
          ((unsigned char *) state)[14] << 16 |
          ((unsigned char *) state)[3] << 24;
  cw[2] =
          ((unsigned char *) state)[8] |
          ((unsigned char *) state)[13] << 8 |
          ((unsigned char *) state)[2] << 16 |
          ((unsigned char *) state)[7] << 24;
  cw[3] =
          ((unsigned char *) state)[12] |
          ((unsigned char *) state)[1] << 8 |
          ((unsigned char *) state)[6] << 16 |
          ((unsigned char *) state)[11] << 24;*//*

  memcpy(state, cb, sizeof(int) * NB);
}*/

__device__ int mul3CUDA(unsigned char dt) {
  int x;
  x = dt << 1;
  if (x & 0x100)
    x = (x ^ 0x1b) & 0xff;
  x ^= dt;

  return (x);
}

__device__ int mul2CUDA(unsigned char dt) {
  int x;
  x = dt << 1;
  if (x & 0x100)
    x = (x ^ 0x1b) & 0xff;

  return (x);
}

__device__ void MixAddRound(int * state, int n){
  int cw[NB], key[NB];
  memcpy(key, &(rkey[n]), sizeof(int) * NB);
  memcpy(cw, state, sizeof(int) * NB);
  cw[0] =(mul2CUDA(((unsigned char *) cw)[0]) ^
          mul3CUDA(((unsigned char *) cw)[1]) ^
          ((unsigned char *) cw)[2] ^
          ((unsigned char *) cw)[3]
          |
          (mul2CUDA(((unsigned char *) cw)[1]) ^
           mul3CUDA(((unsigned char *) cw)[2]) ^
           ((unsigned char *) cw)[3] ^
           ((unsigned char *) cw)[0]) << 8
          |
          (mul2CUDA(((unsigned char *) cw)[2]) ^
           mul3CUDA(((unsigned char *) cw)[3]) ^
           ((unsigned char *) cw)[0] ^
           ((unsigned char *) cw)[1]) << 16
          |
          (mul2CUDA(((unsigned char *) cw)[3]) ^
           mul3CUDA(((unsigned char *) cw)[0]) ^
           ((unsigned char *) cw)[1] ^
           ((unsigned char *) cw)[2]) << 24)
         ^ key[0];

  cw[1] = (mul2CUDA(((unsigned char *) cw)[4]) ^
           mul3CUDA(((unsigned char *) cw)[5]) ^
           ((unsigned char *) cw)[6] ^
           ((unsigned char *) cw)[7]
           |
           (mul2CUDA(((unsigned char *) cw)[5]) ^
            mul3CUDA(((unsigned char *) cw)[6]) ^
            ((unsigned char *) cw)[7] ^
            ((unsigned char *) cw)[4]) << 8
           |
           (mul2CUDA(((unsigned char *) cw)[6]) ^
            mul3CUDA(((unsigned char *) cw)[7]) ^
            ((unsigned char *) cw)[4] ^
            ((unsigned char *) cw)[5]) << 16
           |
           (mul2CUDA(((unsigned char *) cw)[7]) ^
            mul3CUDA(((unsigned char *) cw)[4]) ^
            ((unsigned char *) cw)[5] ^
            ((unsigned char *) cw)[6]) << 24)
          ^ key[1];

  cw[2] = (mul2CUDA(((unsigned char *) cw)[8]) ^
           mul3CUDA(((unsigned char *) cw)[9]) ^
           ((unsigned char *) cw)[10] ^
           ((unsigned char *) cw)[11]
           |
           (mul2CUDA(((unsigned char *) cw)[9]) ^
            mul3CUDA(((unsigned char *) cw)[10]) ^
            ((unsigned char *) cw)[11] ^
            ((unsigned char *) cw)[8]) << 8
           |
           (mul2CUDA(((unsigned char *) cw)[10]) ^
            mul3CUDA(((unsigned char *) cw)[11]) ^
            ((unsigned char *) cw)[8] ^
            ((unsigned char *) cw)[9]) << 16
           |
           (mul2CUDA(((unsigned char *) cw)[11]) ^
            mul3CUDA(((unsigned char *) cw)[8]) ^
            ((unsigned char *) cw)[9] ^
            ((unsigned char *) cw)[10]) << 24)
          ^ key[2];

  cw[3] = (mul2CUDA(((unsigned char *) cw)[12]) ^
           mul3CUDA(((unsigned char *) cw)[13]) ^
           ((unsigned char *) cw)[14] ^
           ((unsigned char *) cw)[15]
           |
           (mul2CUDA(((unsigned char *) cw)[13]) ^
            mul3CUDA(((unsigned char *) cw)[14]) ^
            ((unsigned char *) cw)[15] ^
            ((unsigned char *) cw)[12]) << 8
           |
           (mul2CUDA(((unsigned char *) cw)[14]) ^
            mul3CUDA(((unsigned char *) cw)[15]) ^
            ((unsigned char *) cw)[12] ^
            ((unsigned char *) cw)[13]) << 16
           |
           (mul2CUDA(((unsigned char *) cw)[15]) ^
            mul3CUDA(((unsigned char *) cw)[12]) ^
            ((unsigned char *) cw)[13] ^
            ((unsigned char *) cw)[14]) << 24)
          ^ key[3];
  memcpy(state, cw, sizeof(int) * NB);
}

__device__ void MixColumnsCUDA(int *state) {
  int cw[NB];
  memcpy(cw, state, sizeof(int) * NB);
  cw[0] = mul2CUDA(((unsigned char *) cw)[0]) ^
          mul3CUDA(((unsigned char *) cw)[1]) ^
          ((unsigned char *) cw)[2] ^
          ((unsigned char *) cw)[3]
          |
          (mul2CUDA(((unsigned char *) cw)[1]) ^
           mul3CUDA(((unsigned char *) cw)[2]) ^
           ((unsigned char *) cw)[3] ^
           ((unsigned char *) cw)[0]) << 8
          |
          (mul2CUDA(((unsigned char *) cw)[2]) ^
           mul3CUDA(((unsigned char *) cw)[3]) ^
           ((unsigned char *) cw)[0] ^
           ((unsigned char *) cw)[1]) << 16
          |
          (mul2CUDA(((unsigned char *) cw)[3]) ^
           mul3CUDA(((unsigned char *) cw)[0]) ^
           ((unsigned char *) cw)[1] ^
           ((unsigned char *) cw)[2]) << 24;

  cw[1] = mul2CUDA(((unsigned char *) cw)[4]) ^
          mul3CUDA(((unsigned char *) cw)[5]) ^
          ((unsigned char *) cw)[6] ^
          ((unsigned char *) cw)[7]
          |
          (mul2CUDA(((unsigned char *) cw)[5]) ^
           mul3CUDA(((unsigned char *) cw)[6]) ^
           ((unsigned char *) cw)[7] ^
           ((unsigned char *) cw)[4]) << 8
          |
          (mul2CUDA(((unsigned char *) cw)[6]) ^
           mul3CUDA(((unsigned char *) cw)[7]) ^
           ((unsigned char *) cw)[4] ^
           ((unsigned char *) cw)[5]) << 16
          |
          (mul2CUDA(((unsigned char *) cw)[7]) ^
           mul3CUDA(((unsigned char *) cw)[4]) ^
           ((unsigned char *) cw)[5] ^
           ((unsigned char *) cw)[6]) << 24;

  cw[2] = mul2CUDA(((unsigned char *) cw)[8]) ^
          mul3CUDA(((unsigned char *) cw)[9]) ^
          ((unsigned char *) cw)[10] ^
          ((unsigned char *) cw)[11]
          |
          (mul2CUDA(((unsigned char *) cw)[9]) ^
           mul3CUDA(((unsigned char *) cw)[10]) ^
           ((unsigned char *) cw)[11] ^
           ((unsigned char *) cw)[8]) << 8
          |
          (mul2CUDA(((unsigned char *) cw)[10]) ^
           mul3CUDA(((unsigned char *) cw)[11]) ^
           ((unsigned char *) cw)[8] ^
           ((unsigned char *) cw)[9]) << 16
          |
          (mul2CUDA(((unsigned char *) cw)[11]) ^
           mul3CUDA(((unsigned char *) cw)[8]) ^
           ((unsigned char *) cw)[9] ^
           ((unsigned char *) cw)[10]) << 24;

  cw[3] = mul2CUDA(((unsigned char *) cw)[12]) ^
          mul3CUDA(((unsigned char *) cw)[13]) ^
          ((unsigned char *) cw)[14] ^
          ((unsigned char *) cw)[15]
          |
          (mul2CUDA(((unsigned char *) cw)[13]) ^
           mul3CUDA(((unsigned char *) cw)[14]) ^
           ((unsigned char *) cw)[15] ^
           ((unsigned char *) cw)[12]) << 8
          |
          (mul2CUDA(((unsigned char *) cw)[14]) ^
           mul3CUDA(((unsigned char *) cw)[15]) ^
           ((unsigned char *) cw)[12] ^
           ((unsigned char *) cw)[13]) << 16
          |
          (mul2CUDA(((unsigned char *) cw)[15]) ^
           mul3CUDA(((unsigned char *) cw)[12]) ^
           ((unsigned char *) cw)[13] ^
           ((unsigned char *) cw)[14]) << 24;
  memcpy(state, cw, sizeof(int) * NB);
}


__device__ void AddRoundKeyCUDA(int *state, int *w, int n) {
  int cw[NB], key[NB];
  memcpy(cw, state, sizeof(int) * NB);
  memcpy(key, &(rkey[n]), sizeof(int) * NB);
  cw[0] ^= key[0];
  cw[1] ^= key[1];
  cw[2] ^= key[2];
  cw[3] ^= key[3];
  memcpy(state, cw, sizeof(int) * NB);
}

__device__ void CipherCUDA(int *pt, unsigned char *ct, int *rkey) {
  int rnd, threadId = ((blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
  int* state = pt;
  unsigned char cb[NBb], cb2[NBb];
  int* cw = (int *)cb;
  int* cw2 = (int *)cb2;
//  int state[NB];
//  memcpy(state, pt, sizeof(int) * NB);

  cw2[0] = state[0] ^ rkey[0];
  cw2[1] = state[1] ^ rkey[1];
  cw2[2] = state[2] ^ rkey[2];
  cw2[3] = state[3] ^ rkey[3];

//  round 1
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
           SboxCUDA[((unsigned char *) cw2)[10]] ^
           SboxCUDA[((unsigned char *) cw2)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            SboxCUDA[((unsigned char *) cw2)[15]] ^
            SboxCUDA[((unsigned char *) cw2)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            SboxCUDA[((unsigned char *) cw2)[0]] ^
            SboxCUDA[((unsigned char *) cw2)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
            SboxCUDA[((unsigned char *) cw2)[5]] ^
            SboxCUDA[((unsigned char *) cw2)[10]]) << 24)
          ^ rkey[4];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
           SboxCUDA[((unsigned char *) cw2)[14]] ^
           SboxCUDA[((unsigned char *) cw2)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            SboxCUDA[((unsigned char *) cw2)[3]] ^
            SboxCUDA[((unsigned char *) cw2)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            SboxCUDA[((unsigned char *) cw2)[4]] ^
            SboxCUDA[((unsigned char *) cw2)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
            SboxCUDA[((unsigned char *) cw2)[9]] ^
            SboxCUDA[((unsigned char *) cw2)[14]]) << 24)
          ^ rkey[5];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
           SboxCUDA[((unsigned char *) cw2)[2]] ^
           SboxCUDA[((unsigned char *) cw2)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            SboxCUDA[((unsigned char *) cw2)[7]] ^
            SboxCUDA[((unsigned char *) cw2)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            SboxCUDA[((unsigned char *) cw2)[8]] ^
            SboxCUDA[((unsigned char *) cw2)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
            SboxCUDA[((unsigned char *) cw2)[13]] ^
            SboxCUDA[((unsigned char *) cw2)[2]]) << 24)
          ^ rkey[6];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
           SboxCUDA[((unsigned char *) cw2)[6]] ^
           SboxCUDA[((unsigned char *) cw2)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            SboxCUDA[((unsigned char *) cw2)[11]] ^
            SboxCUDA[((unsigned char *) cw2)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            SboxCUDA[((unsigned char *) cw2)[12]] ^
            SboxCUDA[((unsigned char *) cw2)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
            SboxCUDA[((unsigned char *) cw2)[1]] ^
            SboxCUDA[((unsigned char *) cw2)[6]]) << 24)
          ^ rkey[7];
//  round2
  cw2[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            SboxCUDA[((unsigned char *) cw)[10]] ^
            SboxCUDA[((unsigned char *) cw)[15]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             SboxCUDA[((unsigned char *) cw)[15]] ^
             SboxCUDA[((unsigned char *) cw)[0]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             SboxCUDA[((unsigned char *) cw)[0]] ^
             SboxCUDA[((unsigned char *) cw)[5]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
             SboxCUDA[((unsigned char *) cw)[5]] ^
             SboxCUDA[((unsigned char *) cw)[10]]) << 24)
           ^ rkey[8];

  cw2[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            SboxCUDA[((unsigned char *) cw)[14]] ^
            SboxCUDA[((unsigned char *) cw)[3]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             SboxCUDA[((unsigned char *) cw)[3]] ^
             SboxCUDA[((unsigned char *) cw)[4]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             SboxCUDA[((unsigned char *) cw)[4]] ^
             SboxCUDA[((unsigned char *) cw)[9]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
             SboxCUDA[((unsigned char *) cw)[9]] ^
             SboxCUDA[((unsigned char *) cw)[14]]) << 24)
           ^ rkey[9];

  cw2[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            SboxCUDA[((unsigned char *) cw)[2]] ^
            SboxCUDA[((unsigned char *) cw)[7]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             SboxCUDA[((unsigned char *) cw)[7]] ^
             SboxCUDA[((unsigned char *) cw)[8]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             SboxCUDA[((unsigned char *) cw)[8]] ^
             SboxCUDA[((unsigned char *) cw)[13]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
             SboxCUDA[((unsigned char *) cw)[13]] ^
             SboxCUDA[((unsigned char *) cw)[2]]) << 24)
           ^ rkey[10];

  cw2[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            SboxCUDA[((unsigned char *) cw)[6]] ^
            SboxCUDA[((unsigned char *) cw)[11]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             SboxCUDA[((unsigned char *) cw)[11]] ^
             SboxCUDA[((unsigned char *) cw)[12]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             SboxCUDA[((unsigned char *) cw)[12]] ^
             SboxCUDA[((unsigned char *) cw)[1]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
             SboxCUDA[((unsigned char *) cw)[1]] ^
             SboxCUDA[((unsigned char *) cw)[6]]) << 24)
           ^ rkey[11];

//  round 3
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
           SboxCUDA[((unsigned char *) cw2)[10]] ^
           SboxCUDA[((unsigned char *) cw2)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            SboxCUDA[((unsigned char *) cw2)[15]] ^
            SboxCUDA[((unsigned char *) cw2)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            SboxCUDA[((unsigned char *) cw2)[0]] ^
            SboxCUDA[((unsigned char *) cw2)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
            SboxCUDA[((unsigned char *) cw2)[5]] ^
            SboxCUDA[((unsigned char *) cw2)[10]]) << 24)
          ^ rkey[12];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
           SboxCUDA[((unsigned char *) cw2)[14]] ^
           SboxCUDA[((unsigned char *) cw2)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            SboxCUDA[((unsigned char *) cw2)[3]] ^
            SboxCUDA[((unsigned char *) cw2)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            SboxCUDA[((unsigned char *) cw2)[4]] ^
            SboxCUDA[((unsigned char *) cw2)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
            SboxCUDA[((unsigned char *) cw2)[9]] ^
            SboxCUDA[((unsigned char *) cw2)[14]]) << 24)
          ^ rkey[13];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
           SboxCUDA[((unsigned char *) cw2)[2]] ^
           SboxCUDA[((unsigned char *) cw2)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            SboxCUDA[((unsigned char *) cw2)[7]] ^
            SboxCUDA[((unsigned char *) cw2)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            SboxCUDA[((unsigned char *) cw2)[8]] ^
            SboxCUDA[((unsigned char *) cw2)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
            SboxCUDA[((unsigned char *) cw2)[13]] ^
            SboxCUDA[((unsigned char *) cw2)[2]]) << 24)
          ^ rkey[13];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
           SboxCUDA[((unsigned char *) cw2)[6]] ^
           SboxCUDA[((unsigned char *) cw2)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            SboxCUDA[((unsigned char *) cw2)[11]] ^
            SboxCUDA[((unsigned char *) cw2)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            SboxCUDA[((unsigned char *) cw2)[12]] ^
            SboxCUDA[((unsigned char *) cw2)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
            SboxCUDA[((unsigned char *) cw2)[1]] ^
            SboxCUDA[((unsigned char *) cw2)[6]]) << 24)
          ^ rkey[15];

//  round 4
  cw2[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            SboxCUDA[((unsigned char *) cw)[10]] ^
            SboxCUDA[((unsigned char *) cw)[15]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             SboxCUDA[((unsigned char *) cw)[15]] ^
             SboxCUDA[((unsigned char *) cw)[0]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             SboxCUDA[((unsigned char *) cw)[0]] ^
             SboxCUDA[((unsigned char *) cw)[5]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
             SboxCUDA[((unsigned char *) cw)[5]] ^
             SboxCUDA[((unsigned char *) cw)[10]]) << 24)
           ^ rkey[16];

  cw2[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            SboxCUDA[((unsigned char *) cw)[14]] ^
            SboxCUDA[((unsigned char *) cw)[3]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             SboxCUDA[((unsigned char *) cw)[3]] ^
             SboxCUDA[((unsigned char *) cw)[4]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             SboxCUDA[((unsigned char *) cw)[4]] ^
             SboxCUDA[((unsigned char *) cw)[9]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
             SboxCUDA[((unsigned char *) cw)[9]] ^
             SboxCUDA[((unsigned char *) cw)[14]]) << 24)
           ^ rkey[17];

  cw2[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            SboxCUDA[((unsigned char *) cw)[2]] ^
            SboxCUDA[((unsigned char *) cw)[7]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             SboxCUDA[((unsigned char *) cw)[7]] ^
             SboxCUDA[((unsigned char *) cw)[8]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             SboxCUDA[((unsigned char *) cw)[8]] ^
             SboxCUDA[((unsigned char *) cw)[13]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
             SboxCUDA[((unsigned char *) cw)[13]] ^
             SboxCUDA[((unsigned char *) cw)[2]]) << 24)
           ^ rkey[18];

  cw2[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            SboxCUDA[((unsigned char *) cw)[6]] ^
            SboxCUDA[((unsigned char *) cw)[11]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             SboxCUDA[((unsigned char *) cw)[11]] ^
             SboxCUDA[((unsigned char *) cw)[12]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             SboxCUDA[((unsigned char *) cw)[12]] ^
             SboxCUDA[((unsigned char *) cw)[1]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
             SboxCUDA[((unsigned char *) cw)[1]] ^
             SboxCUDA[((unsigned char *) cw)[6]]) << 24)
           ^ rkey[19];

//  round 5
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
           SboxCUDA[((unsigned char *) cw2)[10]] ^
           SboxCUDA[((unsigned char *) cw2)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            SboxCUDA[((unsigned char *) cw2)[15]] ^
            SboxCUDA[((unsigned char *) cw2)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            SboxCUDA[((unsigned char *) cw2)[0]] ^
            SboxCUDA[((unsigned char *) cw2)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
            SboxCUDA[((unsigned char *) cw2)[5]] ^
            SboxCUDA[((unsigned char *) cw2)[10]]) << 24)
          ^ rkey[20];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
           SboxCUDA[((unsigned char *) cw2)[14]] ^
           SboxCUDA[((unsigned char *) cw2)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            SboxCUDA[((unsigned char *) cw2)[3]] ^
            SboxCUDA[((unsigned char *) cw2)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            SboxCUDA[((unsigned char *) cw2)[4]] ^
            SboxCUDA[((unsigned char *) cw2)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
            SboxCUDA[((unsigned char *) cw2)[9]] ^
            SboxCUDA[((unsigned char *) cw2)[14]]) << 24)
          ^ rkey[21];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
           SboxCUDA[((unsigned char *) cw2)[2]] ^
           SboxCUDA[((unsigned char *) cw2)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            SboxCUDA[((unsigned char *) cw2)[7]] ^
            SboxCUDA[((unsigned char *) cw2)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            SboxCUDA[((unsigned char *) cw2)[8]] ^
            SboxCUDA[((unsigned char *) cw2)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
            SboxCUDA[((unsigned char *) cw2)[13]] ^
            SboxCUDA[((unsigned char *) cw2)[2]]) << 24)
          ^ rkey[22];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
           SboxCUDA[((unsigned char *) cw2)[6]] ^
           SboxCUDA[((unsigned char *) cw2)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            SboxCUDA[((unsigned char *) cw2)[11]] ^
            SboxCUDA[((unsigned char *) cw2)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            SboxCUDA[((unsigned char *) cw2)[12]] ^
            SboxCUDA[((unsigned char *) cw2)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
            SboxCUDA[((unsigned char *) cw2)[1]] ^
            SboxCUDA[((unsigned char *) cw2)[6]]) << 24)
          ^ rkey[23];

//  round 6
  cw2[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            SboxCUDA[((unsigned char *) cw)[10]] ^
            SboxCUDA[((unsigned char *) cw)[15]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             SboxCUDA[((unsigned char *) cw)[15]] ^
             SboxCUDA[((unsigned char *) cw)[0]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             SboxCUDA[((unsigned char *) cw)[0]] ^
             SboxCUDA[((unsigned char *) cw)[5]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
             SboxCUDA[((unsigned char *) cw)[5]] ^
             SboxCUDA[((unsigned char *) cw)[10]]) << 24)
           ^ rkey[24];

  cw2[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            SboxCUDA[((unsigned char *) cw)[14]] ^
            SboxCUDA[((unsigned char *) cw)[3]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             SboxCUDA[((unsigned char *) cw)[3]] ^
             SboxCUDA[((unsigned char *) cw)[4]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             SboxCUDA[((unsigned char *) cw)[4]] ^
             SboxCUDA[((unsigned char *) cw)[9]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
             SboxCUDA[((unsigned char *) cw)[9]] ^
             SboxCUDA[((unsigned char *) cw)[14]]) << 24)
           ^ rkey[25];

  cw2[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            SboxCUDA[((unsigned char *) cw)[2]] ^
            SboxCUDA[((unsigned char *) cw)[7]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             SboxCUDA[((unsigned char *) cw)[7]] ^
             SboxCUDA[((unsigned char *) cw)[8]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             SboxCUDA[((unsigned char *) cw)[8]] ^
             SboxCUDA[((unsigned char *) cw)[13]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
             SboxCUDA[((unsigned char *) cw)[13]] ^
             SboxCUDA[((unsigned char *) cw)[2]]) << 24)
           ^ rkey[26];

  cw2[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            SboxCUDA[((unsigned char *) cw)[6]] ^
            SboxCUDA[((unsigned char *) cw)[11]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             SboxCUDA[((unsigned char *) cw)[11]] ^
             SboxCUDA[((unsigned char *) cw)[12]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             SboxCUDA[((unsigned char *) cw)[12]] ^
             SboxCUDA[((unsigned char *) cw)[1]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
             SboxCUDA[((unsigned char *) cw)[1]] ^
             SboxCUDA[((unsigned char *) cw)[6]]) << 24)
           ^ rkey[27];

  //  round 7
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
           SboxCUDA[((unsigned char *) cw2)[10]] ^
           SboxCUDA[((unsigned char *) cw2)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            SboxCUDA[((unsigned char *) cw2)[15]] ^
            SboxCUDA[((unsigned char *) cw2)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            SboxCUDA[((unsigned char *) cw2)[0]] ^
            SboxCUDA[((unsigned char *) cw2)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
            SboxCUDA[((unsigned char *) cw2)[5]] ^
            SboxCUDA[((unsigned char *) cw2)[10]]) << 24)
          ^ rkey[28];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
           SboxCUDA[((unsigned char *) cw2)[14]] ^
           SboxCUDA[((unsigned char *) cw2)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            SboxCUDA[((unsigned char *) cw2)[3]] ^
            SboxCUDA[((unsigned char *) cw2)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            SboxCUDA[((unsigned char *) cw2)[4]] ^
            SboxCUDA[((unsigned char *) cw2)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
            SboxCUDA[((unsigned char *) cw2)[9]] ^
            SboxCUDA[((unsigned char *) cw2)[14]]) << 24)
          ^ rkey[29];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
           SboxCUDA[((unsigned char *) cw2)[2]] ^
           SboxCUDA[((unsigned char *) cw2)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            SboxCUDA[((unsigned char *) cw2)[7]] ^
            SboxCUDA[((unsigned char *) cw2)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            SboxCUDA[((unsigned char *) cw2)[8]] ^
            SboxCUDA[((unsigned char *) cw2)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
            SboxCUDA[((unsigned char *) cw2)[13]] ^
            SboxCUDA[((unsigned char *) cw2)[2]]) << 24)
          ^ rkey[30];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
           SboxCUDA[((unsigned char *) cw2)[6]] ^
           SboxCUDA[((unsigned char *) cw2)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            SboxCUDA[((unsigned char *) cw2)[11]] ^
            SboxCUDA[((unsigned char *) cw2)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            SboxCUDA[((unsigned char *) cw2)[12]] ^
            SboxCUDA[((unsigned char *) cw2)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
            SboxCUDA[((unsigned char *) cw2)[1]] ^
            SboxCUDA[((unsigned char *) cw2)[6]]) << 24)
          ^ rkey[31];

//  round 8
  cw2[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
            SboxCUDA[((unsigned char *) cw)[10]] ^
            SboxCUDA[((unsigned char *) cw)[15]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[5]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             SboxCUDA[((unsigned char *) cw)[15]] ^
             SboxCUDA[((unsigned char *) cw)[0]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[10]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             SboxCUDA[((unsigned char *) cw)[0]] ^
             SboxCUDA[((unsigned char *) cw)[5]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[15]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[0]]) ^
             SboxCUDA[((unsigned char *) cw)[5]] ^
             SboxCUDA[((unsigned char *) cw)[10]]) << 24)
           ^ rkey[32];

  cw2[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
            SboxCUDA[((unsigned char *) cw)[14]] ^
            SboxCUDA[((unsigned char *) cw)[3]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[9]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             SboxCUDA[((unsigned char *) cw)[3]] ^
             SboxCUDA[((unsigned char *) cw)[4]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[14]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             SboxCUDA[((unsigned char *) cw)[4]] ^
             SboxCUDA[((unsigned char *) cw)[9]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[3]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[4]]) ^
             SboxCUDA[((unsigned char *) cw)[9]] ^
             SboxCUDA[((unsigned char *) cw)[14]]) << 24)
           ^ rkey[33];

  cw2[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
            SboxCUDA[((unsigned char *) cw)[2]] ^
            SboxCUDA[((unsigned char *) cw)[7]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[13]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             SboxCUDA[((unsigned char *) cw)[7]] ^
             SboxCUDA[((unsigned char *) cw)[8]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[2]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             SboxCUDA[((unsigned char *) cw)[8]] ^
             SboxCUDA[((unsigned char *) cw)[13]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[7]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[8]]) ^
             SboxCUDA[((unsigned char *) cw)[13]] ^
             SboxCUDA[((unsigned char *) cw)[2]]) << 24)
           ^ rkey[34];

  cw2[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
            SboxCUDA[((unsigned char *) cw)[6]] ^
            SboxCUDA[((unsigned char *) cw)[11]]
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[1]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             SboxCUDA[((unsigned char *) cw)[11]] ^
             SboxCUDA[((unsigned char *) cw)[12]]) << 8
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[6]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             SboxCUDA[((unsigned char *) cw)[12]] ^
             SboxCUDA[((unsigned char *) cw)[1]]) << 16
            |
            (mul2CUDA(SboxCUDA[((unsigned char *) cw)[11]]) ^
             mul3CUDA(SboxCUDA[((unsigned char *) cw)[12]]) ^
             SboxCUDA[((unsigned char *) cw)[1]] ^
             SboxCUDA[((unsigned char *) cw)[6]]) << 24)
           ^ rkey[35];

  //  round 9
  cw[0] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
           SboxCUDA[((unsigned char *) cw2)[10]] ^
           SboxCUDA[((unsigned char *) cw2)[15]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[5]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            SboxCUDA[((unsigned char *) cw2)[15]] ^
            SboxCUDA[((unsigned char *) cw2)[0]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[10]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            SboxCUDA[((unsigned char *) cw2)[0]] ^
            SboxCUDA[((unsigned char *) cw2)[5]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[15]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[0]]) ^
            SboxCUDA[((unsigned char *) cw2)[5]] ^
            SboxCUDA[((unsigned char *) cw2)[10]]) << 24)
          ^ rkey[36];

  cw[1] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
           SboxCUDA[((unsigned char *) cw2)[14]] ^
           SboxCUDA[((unsigned char *) cw2)[3]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[9]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            SboxCUDA[((unsigned char *) cw2)[3]] ^
            SboxCUDA[((unsigned char *) cw2)[4]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[14]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            SboxCUDA[((unsigned char *) cw2)[4]] ^
            SboxCUDA[((unsigned char *) cw2)[9]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[3]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[4]]) ^
            SboxCUDA[((unsigned char *) cw2)[9]] ^
            SboxCUDA[((unsigned char *) cw2)[14]]) << 24)
          ^ rkey[37];

  cw[2] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
           SboxCUDA[((unsigned char *) cw2)[2]] ^
           SboxCUDA[((unsigned char *) cw2)[7]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[13]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            SboxCUDA[((unsigned char *) cw2)[7]] ^
            SboxCUDA[((unsigned char *) cw2)[8]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[2]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            SboxCUDA[((unsigned char *) cw2)[8]] ^
            SboxCUDA[((unsigned char *) cw2)[13]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[7]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[8]]) ^
            SboxCUDA[((unsigned char *) cw2)[13]] ^
            SboxCUDA[((unsigned char *) cw2)[2]]) << 24)
          ^ rkey[38];

  cw[3] = (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
           mul3CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
           SboxCUDA[((unsigned char *) cw2)[6]] ^
           SboxCUDA[((unsigned char *) cw2)[11]]
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[1]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            SboxCUDA[((unsigned char *) cw2)[11]] ^
            SboxCUDA[((unsigned char *) cw2)[12]]) << 8
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[6]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            SboxCUDA[((unsigned char *) cw2)[12]] ^
            SboxCUDA[((unsigned char *) cw2)[1]]) << 16
           |
           (mul2CUDA(SboxCUDA[((unsigned char *) cw2)[11]]) ^
            mul3CUDA(SboxCUDA[((unsigned char *) cw2)[12]]) ^
            SboxCUDA[((unsigned char *) cw2)[1]] ^
            SboxCUDA[((unsigned char *) cw2)[6]]) << 24)
          ^ rkey[39];

  cb2[0] = SboxCUDA[cb[0]];
  cb2[1] = SboxCUDA[cb[5]];
  cb2[2] = SboxCUDA[cb[10]];
  cb2[3] = SboxCUDA[cb[15]];
  cw2[0] ^= rkey[40];
  cb2[4] = SboxCUDA[cb[4]];
  cb2[5] = SboxCUDA[cb[9]];
  cb2[6] = SboxCUDA[cb[14]];
  cb2[7] = SboxCUDA[cb[3]];
  cw2[1] ^= rkey[41];
  cb2[8] = SboxCUDA[cb[8]];
  cb2[9] = SboxCUDA[cb[13]];
  cb2[10] = SboxCUDA[cb[2]];
  cb2[11] = SboxCUDA[cb[7]];
  cw2[2] ^= rkey[42];
  cb2[12] = SboxCUDA[cb[12]];
  cb2[13] = SboxCUDA[cb[1]];
  cb2[14] = SboxCUDA[cb[6]];
  cb2[15] = SboxCUDA[cb[11]];
  cw2[3] ^= rkey[43];
  if (threadId == 0 ) {
    printf("cw0: 0x%x\n", cw[0]);
    printf("cw1: 0x%x\n", cw[1]);
    printf("cw2: 0x%x\n", cw[2]);
    printf("cw3: 0x%x\n", cw[3]);
  }
//  SubShift(cw);
//  SubBytesCUDA(state);
//  ShiftRowsCUDA(state);
//  AddRoundKeyCUDA(cw, rkey, rnd);
  memcpy(&ct[(((blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x) << 4], cb2,
         sizeof(int) * NB);

  return;
}

__global__ void device_aes_encrypt(unsigned char *pt, unsigned char *ct, long int size) {

  //This kernel executes AES encryption on a GPU.
  //Please modify this kernel!!
  int thread_id = ((blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

 /* if (thread_id == 0)
    printf("size = %ld\n", size);
//  printf("You can use printf function to eliminate bugs in your kernel.\n");
*/
  __shared__ int state[BLOCKSIZE][NB];
  memcpy(&(state[threadIdx.x][0]), &(pt[thread_id << 4]), sizeof(unsigned char) * NBb);
  CipherCUDA(&(state[threadIdx.x][0]), ct, rkey);
//  memcpy(&ct[thread_id << 4], &state[threadIdx.x], sizeof(unsigned char) * NBb);
}

void launch_aes_kernel(unsigned char *pt, int *rk, unsigned char *ct, long int size) {

  //This function launches the AES kernel.
  //Please modify this function for AES kernel.
  //In this function, you need to allocate the device memory and so on.
  unsigned char *d_pt, *d_ct;

  dim3 dim_grid(GRIDSIZE_X, GRIDSIZE_Y, GRIDSIZE_Z), dim_block(BLOCKSIZE, 1, 1);

  hipMalloc((void **) &d_pt, sizeof(unsigned char) * size);
//  hipMalloc((void **) &d_rkey, sizeof(int) * 44);
  hipMalloc((void **) &d_ct, sizeof(unsigned char) * size);

  hipMemset(d_pt, 0, sizeof(unsigned char) * size);
  hipMemcpy(d_pt, pt, sizeof(unsigned char) * size, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(rkey), rk, sizeof(int) * 44);
//  hipMemcpyToSymbol(HIP_SYMBOL(state_org), pt, sizeof(unsigned char) * size);

  device_aes_encrypt <<< dim_grid, dim_block >>> (d_pt, d_ct, size);

  hipMemcpy(ct, d_ct, sizeof(unsigned char) * size, hipMemcpyDeviceToHost);

  hipFree(d_pt);
  hipFree(d_ct);
}












